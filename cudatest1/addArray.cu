#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<ctime>

using namespace std;

__global__ void AddInts(int *a, int *b,int count) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id < count) {
		a[id] += b[id];
	}
}

int main() {
	int count = 1000;
	srand(time(NULL));
	int *h_a = new int[count];
	int *h_b = new int[count];
	for (int i = 0;  i < count; i++) {
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;
	}
	cout << "prior to addition:" << endl;
	for (int i = 0; i < 5; i++) {
		cout << h_a[i] << " " << h_b[i] << endl;
	}

	int *d_a, *d_b;
	hipMalloc(&d_a, sizeof(int)*count);
	hipMalloc(&d_b, sizeof(int)*count);
	hipMemcpy(d_a, h_a, sizeof(int)*count, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int)*count, hipMemcpyHostToDevice);

	AddInts << <count / 256 + 1, 256 >> > (d_a, d_b, count);
	hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost);
	for (int i = 0; i < 5; i++) {
		cout << "It is" << h_a[i] << endl;
	}
	hipFree(d_a);
	hipFree(d_b);
	delete[] h_a;
	delete[] h_b;

	return 0;

}