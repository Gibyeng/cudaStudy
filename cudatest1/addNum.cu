#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void AddIntsCUDA(int* a, int* b) {
	for (int i = 0; i < 1000005; i++) {
		a[0] += b[0];
	}
}

int main() {
	int a = 0;
	int b = 1;
	int *d_a, *d_b;
	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));
	
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCUDA << <1, 1 >> > (d_a, d_b);
	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
	cout << "The anwer is " << a << endl;
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}